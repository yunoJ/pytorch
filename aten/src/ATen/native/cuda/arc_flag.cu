#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <sys/time.h>
#include <mutex>

#include <ATen/native/cuda/arc_flag.h>
#include <ATen/Context.h>
#define NUM_TENSOR 2048

// [JS] P2P define
#include <queue>
#include <ATen/cuda/CUDAEvent.h>

// Half precision
#include <hip/hip_fp16.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>

#define find(n) (32 * (unsigned int)(n / 1024) + (n % 32))
#define mask(n) (0x80000000 >> (unsigned int)((n % 1024) / 32))

// [JS] Arcp2p Setting flag define
#define ARC_FLAG_VDNN  (1U << 0)
#define ARC_FLAG_FP16  (1U << 1)
#define ARC_FLAG_CSR   (1U << 2)
#define ARC_FLAG_SSD   (1U << 3)
#define ARC_FLAG_TESLA (1U << 4)
#define ARC_FLAG_RAID0 (1U << 5)
#define ARC_FLAG_DEBUG (1U << 6)
// [JS] 7~11 bit will be used for arc_vm (device) cudamalloc size
#define ARC_MEMSIZE_MASK  (0x00000F80)
// 12~16 bit will be used for arc_vm (p2p) cudamalloc size
#define ARC_P2PSIZE_MASK  (0x0001F000)
#define ARC_MEMSIZE_SHIFT (7)

using namespace at::cuda;
__global__ void double_scale(__half *din, double *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize)  dout[idx] = (double)__half2float(din[idx]);
}

__global__ void float_scale(__half *din, float *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize)  dout[idx] = __half2float(din[idx]);
}

__global__ void zero_insert_double(unsigned int *bit, unsigned int *nz_pos, float* din, double *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize) {
    int count = -1;
    if ((unsigned int)(bit[find(idx)] & mask(idx)) > 0) {
      for (int i = (int)(idx / 32) * 32; i < idx + 1; i++) {
        unsigned int mask = bit[find(i)] & mask(i);
        if (mask > 0)  count += 1;
      }
    }

    if (count == -1)  dout[idx] = 0.0;
    else {
      if ((unsigned int)(idx / 32) == 0) {
        dout[idx] = (double)din[count + 0];
      } else {
        dout[idx] = (double)din[count + nz_pos[(unsigned int)(idx / 32) - 1]];
      }
    }
  }
}

__global__ void zero_insert_float(unsigned int *bit, unsigned int *nz_pos, float* din, float *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize) {
    int count = -1;
    if ((unsigned int)(bit[find(idx)] & mask(idx)) > 0) {
      for (int i = (int)(idx / 32) * 32; i < idx + 1; i++) {
        unsigned int mask = bit[find(i)] & mask(i);
        if (mask > 0)  count += 1;
      }
    }

    if (count == -1)  dout[idx] = 0.0f;
    else {
      if ((unsigned int)(idx / 32) == 0) {
        dout[idx] = din[count + 0];
      } else {
        dout[idx] = din[count + nz_pos[(unsigned int)(idx / 32) - 1]];
      }
    }
  }
}

namespace at { namespace native {

using namespace at::cuda;

ARC_memory arc_vm;

typedef struct {
  uint64_t addr;
  uint64_t size;
  uint64_t offs;
  arcp2p_dir dir;
  arcp2p_cpl *p_cpl;

  c10::Storage *stor;
  arcp2p_info *info;
  hipStream_t str;

  // Additional information for post task
  // for GPU to SSD case,
  //  - We need to keep GPU memory until transfer is done.
  //  - Release right after process is completed
  //  - Therefore, we need to keep "c10::Storage" class at here
  // for SSD to GPU case,
  //  - We need to keep required data for half-to-float conversion
  //  - These are only for FP16 and CSR case
} req_element;

std::queue<req_element> req_queue;

ARC_memory::ARC_memory(): relu_thru(false), mapping(false),
    isVDNN(false), isFP16(false), isCSR(false), isUsingSSD(false), isTesla(false), isDebug(false),
    deviceStartBlk(0), p2pStartBlk(0), device_sz(0), max_device(0), p2p_sz(0), max_p2p(0) {
  fp16_ptr_arr = new uint64_t[NUM_TENSOR];
  bit_ptr_arr = new uint64_t[NUM_TENSOR];
  pos_ptr_arr = new uint64_t[NUM_TENSOR];
  resize_arr = new int[NUM_TENSOR];
  numel_arr = new size_t[NUM_TENSOR];
  elem_arr = new int[NUM_TENSOR];
  cpl_flu_ptr_arr = new uint64_t[NUM_TENSOR];
  cpl_pre_ptr_arr = new uint64_t[NUM_TENSOR];
  offset_arr = new uint64_t[NUM_TENSOR];
  dir_arr = new arcp2p_dir[NUM_TENSOR];

  event_arr_d2h = new bool[NUM_TENSOR];
  event_arr_h2d = new bool[NUM_TENSOR];

  for(int i = 0; i < NUM_TENSOR; i++) {
    event_arr_d2h[i] = false;
    event_arr_h2d[i] = false;
  }

  memset(dir_arr, 0, sizeof(arcp2p_dir) * NUM_TENSOR);
}

ARC_memory::~ARC_memory() {
  if (device_sz > 0) {
    hipFree(deviceAddr);
    delete[] deviceTable;
    delete[] device_page_map;
  }

  if (p2p_sz > 0) {
    hipFree(p2pAddr);
    delete[] p2pTable;
    delete[] p2p_page_map;
  }

  delete[] fp16_ptr_arr;
  delete[] bit_ptr_arr;
  delete[] pos_ptr_arr;
  delete[] resize_arr;
  delete[] numel_arr;
  delete[] elem_arr;
  delete[] cpl_flu_ptr_arr;
  delete[] cpl_pre_ptr_arr;
  delete[] offset_arr;
  delete[] dir_arr;

  delete[] event_arr_d2h;
  delete[] event_arr_h2d;

  if (true == isUsingSSD)
  {
    arcp2p_synchronize(arc_handle);
    if (true == isTesla)
    {
      arcp2p_bar_detach(arc_handle);
    }
    arcp2p_release(arc_handle);
  }
}

void ARC_memory::device_malloc(void** gpu_ptr, size_t size) {
  int reqBlk = std::ceil((double)size / (double)BLK_SZ);
  int blkCheck = 0;
  int retryCnt = 0;

  if (device_sz == 0) return;

  if (reqBlk == 0) return;

  dev.lock();

  while (true) {
    for (int i = deviceStartBlk; i < max_device; i++) {
      blkCheck += 1;
      if (deviceTable[i]) {
        if (device_page_map[i] == 0) {
          LOG(FATAL) << "device_page_map[" << i << "] is zero, size: " << size << ", " << blkCheck << ", " << reqBlk;
        }
        i += device_page_map[i] - 1;
        blkCheck = 0; deviceStartBlk = i + 1;
        continue;
      }

      if (blkCheck == reqBlk) {
        device_page_map[deviceStartBlk] = reqBlk;
        *gpu_ptr = (void* )((size_t)deviceAddr + (deviceStartBlk * BLK_SZ));

        for (int i = deviceStartBlk; i < deviceStartBlk + reqBlk; i++) {
          deviceTable[i] = true;
        }

        deviceStartBlk += reqBlk;

        dev.unlock();
        return;
      }
    }
    deviceStartBlk = 0;  blkCheck = 0;

    if (retryCnt++ > 2) {
      dev.unlock();
      *gpu_ptr = NULL;
      return;
    }
  }
}

void ARC_memory::device_free(void* addr, size_t size) {
  unsigned int startBlk = ((size_t)addr - (size_t)deviceAddr) / BLK_SZ;
  unsigned int reqBlk = std::ceil((double)size / (double)BLK_SZ);

  if (device_sz == 0) return;

  dev.lock();

  device_page_map[startBlk] = 0;

  for (unsigned int i = startBlk; i < startBlk + reqBlk; i++) {
    deviceTable[i] = false;
  }

  deviceStartBlk = std::min((unsigned int)deviceStartBlk, startBlk);

  dev.unlock();
}

int ARC_memory::device_occupancy() {
  int occupancy;
  dev.lock();

  for(int i = 0; i < max_device; i++) {
    occupancy += device_page_map[i];
  }

  dev.unlock();
  return occupancy;
}

void ARC_memory::p2p_malloc(void** gpu_ptr, size_t size) {
  int reqBlk = std::ceil((double)size / (double)BLK_SZ);
  int blkCheck = 0;
  int retryCnt = 0;

  if (p2p_sz == 0) return;

  if (reqBlk == 0) return;

  p2p.lock();

  while (true) {
    for (int i = p2pStartBlk; i < max_p2p; i++) {
      blkCheck += 1;
      if (p2pTable[i]) {
        if (p2p_page_map[i] == 0) {
          LOG(FATAL) << "p2p_page_map[" << i << "] is zero, size: " << size << ", " << blkCheck << ", " << reqBlk;
        }
        i += p2p_page_map[i] - 1;
        blkCheck = 0; p2pStartBlk = i + 1;
        continue;
      }

      if (blkCheck == reqBlk) {
        p2p_page_map[p2pStartBlk] = reqBlk;
        *gpu_ptr = (void* )((size_t)p2pAddr + (p2pStartBlk * BLK_SZ));

        for (int i = p2pStartBlk; i < p2pStartBlk + reqBlk; i++) {
          p2pTable[i] = true;
        }

        p2pStartBlk += reqBlk;

        p2p.unlock();
        return;
      }
    }
    p2pStartBlk = 0;  blkCheck = 0;

    if (retryCnt++ > 2) {
      *gpu_ptr = NULL;
      return;
    }
  }
}

void ARC_memory::p2p_free(void* addr, size_t size) {
  unsigned int startBlk = ((size_t)addr - (size_t)p2pAddr) / BLK_SZ;
  unsigned int reqBlk = std::ceil((double)size / (double)BLK_SZ);

  if (p2p_sz == 0) return;

  p2p.lock();

  p2p_page_map[startBlk] = 0;

  for (unsigned int i = startBlk; i < startBlk + reqBlk; i++) {
    p2pTable[i] = false;
  }

  p2pStartBlk = std::min((unsigned int)p2pStartBlk, startBlk);

  p2p.unlock();
}

void* ARC_memory::get_fp16_addr(int tid) {
  return (void *)fp16_ptr_arr[tid];
}

void ARC_memory::set_fp16_addr(int tid, uint64_t addr) {
  fp16_ptr_arr[tid] = addr;
}

void* ARC_memory::get_device_addr() {
  return p2pAddr;
}

uint64_t ARC_memory::get_device_sz() {
  return p2p_sz;
}

void* ARC_memory::get_bit_addr(int tid) {
  return (void *)bit_ptr_arr[tid];
}

void ARC_memory::set_bit_addr(int tid, uint64_t addr) {
  bit_ptr_arr[tid] = addr;
}

void* ARC_memory::get_pos_addr(int tid) {
  return (void *)pos_ptr_arr[tid];
}

void ARC_memory::set_pos_addr(int tid, uint64_t addr) {
  pos_ptr_arr[tid] = addr;
}

int ARC_memory::get_resize(int tid) {
  return resize_arr[tid];
}

void ARC_memory::set_resize(int tid, int resize) {
  resize_arr[tid] = resize;
}

size_t ARC_memory::get_numel(int tid) {
  return numel_arr[tid];
}

void ARC_memory::set_numel(int tid, size_t numel) {
  numel_arr[tid] = numel;
}

int ARC_memory::get_elem(int tid) {
  return elem_arr[tid];
}

void ARC_memory::set_elem(int tid, int elem) {
  elem_arr[tid] = elem;
}

void* ARC_memory::get_cpl_addr(int tid, arcp2p_dir dir) {
  if (arcp2p_gputossd == dir) {
    return (void *)cpl_flu_ptr_arr[tid];
  } else if (arcp2p_ssdtogpu == dir) {
    return (void *)cpl_pre_ptr_arr[tid];
  } else {
    return nullptr;
  }
}

void ARC_memory::set_cpl_addr(int tid, arcp2p_dir dir, void *addr) {
  if (arcp2p_gputossd == dir) {
    cpl_flu_ptr_arr[tid] = (uint64_t)addr;
  } else if (arcp2p_ssdtogpu == dir) {
    cpl_pre_ptr_arr[tid] = (uint64_t)addr;
  }
}

uint64_t* ARC_memory::get_offset_ptr(int tid) {
  return &offset_arr[tid];
}

arcp2p_dir ARC_memory::get_dir(int tid) {
  return dir_arr[tid];
}

void ARC_memory::set_dir(int tid, arcp2p_dir dir) {
  dir_arr[tid] = dir;
}

bool ARC_memory::is_vdnn(void) {
  return isVDNN;
}

bool ARC_memory::is_fp16(void) {
  return isFP16;
}

bool ARC_memory::is_csr(void) {
  return isCSR;
}

bool ARC_memory::is_using_ssd(void) {
  return isUsingSSD;
}

bool ARC_memory::is_debug(void) {
  return isDebug;
}

void ARC_memory::Arcp2pSetting(int flags) {
  printf("Arcp2pSetting : 0x%x\n", flags);

  uint64_t device_in_gb;
  device_in_gb = (flags & ARC_MEMSIZE_MASK) >> ARC_MEMSIZE_SHIFT;
  device_sz = device_in_gb << 30;
  max_device = device_sz / BLK_SZ;

  uint64_t p2p_in_gb;
  p2p_in_gb = (flags & ARC_P2PSIZE_MASK) >> 12;
  p2p_sz = p2p_in_gb << 30;
  max_p2p = p2p_sz / BLK_SZ;

  printf("Device memory size = %ld GB\n", device_in_gb);
  printf("P2P memory size = %ld GB\n", p2p_in_gb);

  if (device_in_gb > 0) {
    hipMalloc(&deviceAddr, device_sz);
    deviceTable = new bool[max_device];
    memset(deviceTable, 0, sizeof(bool) * max_device);

    device_page_map = new unsigned int[max_device];
    for (int i = 0; i < max_device; i++) {
      device_page_map[i] = 0;
    }
  }

  if (p2p_in_gb > 0) {
    hipMalloc(&p2pAddr, p2p_sz);
    p2pTable = new bool[max_p2p];
    memset(p2pTable, 0, sizeof(bool) * max_p2p);

    p2p_page_map = new unsigned int[max_p2p];
    for (int i = 0; i < max_p2p; i++) {
      p2p_page_map[i] = 0;
    }
  }

  if (flags & ARC_FLAG_VDNN) {
    printf("vDNN flag set\n");
    isVDNN = true;
  }

  if (flags & ARC_FLAG_FP16) {
    printf("FP16 flag set\n");
    isVDNN = true;
    isFP16 = true;
  }

  if (flags & ARC_FLAG_CSR) {
    printf("CSR flag set\n");
    isVDNN = true;
    isFP16 = true;
    isCSR = true;
  }

  if (flags & ARC_FLAG_TESLA) {
    printf("Tesla flag set\n");
    isTesla = true;
  }

  if (flags & ARC_FLAG_SSD) {
    printf("SSD flag set\n");
    // [JS] P2P
    isVDNN = true;
    isUsingSSD = true;
    last_allocated_offset = 0;

    const char *nvme_path_tesla[PATH_LENGTH] = {"0000:65:00.00", "0000:66:00.00"}; // TESLA
    const char *nvme_path_quadro[PATH_LENGTH] = {"0000:85:00.00", ""}; // QUADRO
    const int nvme_cnt = (flags & ARC_FLAG_RAID0)?2:1;

    printf("RAID0 flag check, device cnt %d\n", nvme_cnt);

    void* lib_handle;
    if (!(lib_handle = dlopen("/usr/local/lib/libarcp2p.so", RTLD_LAZY))) {
      fprintf(stderr, "%s\n", dlerror());
      return;
    }

    arcp2p_initialize = (arcp2p_type1_fn)dlsym(lib_handle, "ARCP2P_initialize");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    arcp2p_release    = (arcp2p_type2_fn)dlsym(lib_handle, "ARCP2P_release");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    arcp2p_bar_attach = (arcp2p_type3_fn)dlsym(lib_handle, "ARCP2P_bar_attach");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    arcp2p_bar_detach = (arcp2p_type2_fn)dlsym(lib_handle, "ARCP2P_bar_detach");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    arcp2p_transfer   = (arcp2p_type4_fn)dlsym(lib_handle, "ARCP2P_transfer");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    arcp2p_completion = (arcp2p_type2_fn)dlsym(lib_handle, "ARCP2P_completion");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    arcp2p_synchronize = (arcp2p_type2_fn)dlsym(lib_handle, "ARCP2P_synchronize");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    if (true == isTesla) {
      arc_handle = arcp2p_initialize(nvme_path_tesla, nvme_cnt);
    } else {
      arc_handle = arcp2p_initialize(nvme_path_quadro, nvme_cnt);
    }
  } else { // if not ssd
    isUsingSSD = false;
  }

  if (flags & ARC_FLAG_DEBUG) {
    printf("Debug mode on\n");
    isDebug = true;
    at::globalContext().ARCGlobal.turnOnDebugMode();
  } else {
    isDebug = false;
  }
}

// bar attach
int  ARC_memory::Arcp2pBarMapping(uint64_t addr, uint64_t size) {
  return arcp2p_bar_attach(arc_handle, addr, size);
}

// submission
void ARC_memory::Arcp2pSubmission(uint64_t addr, uint64_t size, uint64_t *p_offs,
    arcp2p_cpl *p_cpl, arcp2p_dir dir, c10::Storage *stor, arcp2p_info *info, hipStream_t str) {
  uint64_t offset, aligned_size;

  const uint64_t prp_align_size = (1UL << 12);
  const uint64_t prp_align_mask = (prp_align_size - 1);

  // align up the size value
  if (size & prp_align_mask) {
    aligned_size = (size + prp_align_size - 1) & (~prp_align_mask);
  } else {
    aligned_size = size;
  }

  if (arcp2p_gputossd == dir) {
    // flush case, need to allocate nvme area
    offset = last_allocated_offset;
    last_allocated_offset = last_allocated_offset + aligned_size;

    *p_offs = offset;
  } else {
    // prefetch case, handle requested nvme offset
    offset = *p_offs;
  }

  req_element req;
  req.addr = addr;
  req.size = aligned_size;
  req.dir = dir;
  req.stor = stor;
  req.info = info;
  req.str = str;

  req.offs = offset;
  req.p_cpl = p_cpl;

  req.p_cpl->requested = true;
  req.p_cpl->arc_handle = arc_handle;

  if (true == isTesla) {
    // directly deliver transfer request to arcp2p library, only for tesla
    arcp2p_transfer(arc_handle, addr, offset, aligned_size, req.p_cpl, dir);
  } else {
    // for quadro, we need to attach bar range before transfer
    // check that queue is empty, else case will be handled at completion function
    if (req_queue.empty()) {
      printf("Transfer directly\n");
      //arcp2p_bar_attach(arc_handle, addr, size);
      // debug code. retry 10 times
      int retrycnt = 0;
      while(ARCP2P_NO_ERROR != arcp2p_bar_attach(arc_handle, addr, size)) {
        retrycnt ++;
        printf("Bar attach failed, retry %d/10\n", retrycnt);
        if (retrycnt >= 10) {
          break;
        }
        arcp2p_bar_detach(arc_handle);
      }
      arcp2p_transfer(arc_handle, addr, offset, aligned_size, req.p_cpl, dir);
    }
  }

  req_queue.push(req);
}

// completion
void ARC_memory::Arcp2pCompletion(bool prefCall) {

  // Automatically prefetch
  if(!at::globalContext().ARCGlobal.isOnDemand()) {
    if (pref_end >= pref_idx && prefCall) {
      if (isDebug) {
        std::cout << "Prefetching oid call: " << pref_it[pref_idx] << std::endl;
      }
      torch::autograd::ARCCppEngine::preFetchAsync(pref_it[pref_idx]);
      pref_idx++;
    }
  }

  if(isUsingSSD) {
    // if req_list empty, nothing to do
    if (req_queue.empty()) {
      return;
    }

    m2.lock();
    // first, run completer of arcp2p, this will update cpl.issued
    arcp2p_completion(arc_handle);

    // we only concern command completion sequentially
    req_element req = req_queue.front();

    if (true == req.p_cpl->issued) {
      // if completed request is ssdtogpu
      // 1. we need to update fetch_loc
      // 2. we should remove loc_element
  
      if (arcp2p_gputossd == req.dir) {
        size_t numel = get_numel(req.info->tid);
        int resize = get_resize(req.info->tid);
        if (isFP16 && (resize > 0)) {
          if (isDebug)
            std::cout << "CSR FP16 mem free tid: " << req.info->tid << ", size: " << sizeof(__half) * resize << ", fp16: " << req.info->ptr << std::endl;
  
          device_free(req.info->ptr, sizeof(__half) * resize);
        } else if (isFP16 && (resize == 0)) {
          if (isDebug)
            std::cout << "No CSR FP16 mem free tid: " << req.info->tid << ", size: " << sizeof(__half) * numel << ", fp16: " << req.info->ptr << std::endl;
  
          device_free(req.info->ptr, sizeof(__half) * numel);
        } else {
          if (isDebug)
            std::cout << "TODO: Duplicated FP16 mem free tid: " << req.info->tid << ", size: " << req.size << ", fp16: " << req.info->ptr << std::endl;
  
          device_free(req.info->ptr, req.size);
        }
  
        event_arr_d2h[req.info->tid] = false;
        delete req.info;
  
        if (false == isFP16)
          delete req.stor;
  
      } else if (arcp2p_ssdtogpu == req.dir) {
        // [TODO] backend job needed for read done case (ex. notify backward operation that data is ready)
        // [TODO] arcp2p_data would be freed here? or after?
  
        // FP16 & CSR handling
        int resize = get_resize(req.info->tid);
  
        if (isFP16 && (resize > 0)) {
          uint64_t nTPB = req.info->ntpb;
          uint64_t numel = req.info->numel;
  
          size_t bit_elements, pos_elements, pos_elements_before;
          bit_elements = (size_t)((numel + 1024 - 1) / 1024) * 32;
          pos_elements_before = (size_t)((numel + 32 - 1) / 32);
          int count = 0;
          while (pos_elements_before != 0) {
            pos_elements_before = pos_elements_before >> 1;  count++;
          }
          pos_elements = 1 << count;
          
          void* bit = arc_vm.get_bit_addr(req.info->tid);
          void* pos = arc_vm.get_pos_addr(req.info->tid);
          float *nz_dst;
          device_malloc((void **)&nz_dst, resize * sizeof(float));
          hipMemsetAsync((void *)nz_dst, 0, resize * sizeof(float), req.str);
  
          float_scale<<<(numel + nTPB - 1) / nTPB, nTPB, 0, req.str>>>((__half *)req.info->ptr, nz_dst, resize);
  
          if (arc_vm.get_elem(req.info->tid) == 8) {
            zero_insert_double<<<(numel + nTPB - 1) / nTPB, nTPB, 0, req.str>>>((unsigned int*)bit, (unsigned int*)pos, nz_dst, (double *)req.info->dst, numel);
          } else {
            zero_insert_float<<<(numel + nTPB - 1) / nTPB, nTPB, 0, req.str>>>((unsigned int*)bit, (unsigned int*)pos, nz_dst, (float *)req.info->dst, numel);
          }
  
          device_free((void *)nz_dst, resize * sizeof(float));
        } else if (isFP16 && (resize == 0)) {
          uint64_t nTPB = req.info->ntpb;
          uint64_t numel = req.info->numel;
  
          if (arc_vm.get_elem(req.info->tid) == 8) {
            double_scale<<<(numel + nTPB - 1) / nTPB, nTPB, 0, req.str>>>((__half* )req.info->ptr, (double* )req.info->dst, numel);
          } else {
            float_scale<<<(numel + nTPB - 1) / nTPB, nTPB, 0, req.str>>>((__half* )req.info->ptr, (float* )req.info->dst, numel);
          }
        } else {
          hipMemcpyAsync(req.info->dst, req.info->ptr, req.size, hipMemcpyDeviceToDevice, req.str);
        }
  
        event_arr_h2d[req.info->tid] = false;
        delete req.info;
      }
  
      req.p_cpl->requested = false;
  
      // remove current element
      req_queue.pop();
  
      if (false == isTesla) {
        arcp2p_bar_detach(arc_handle);
  
        // check if next event is pending
        if (!req_queue.empty()) {
          req = req_queue.front();
          printf("schedule next one. quadro only\n");
          //arcp2p_bar_attach(arc_handle, req.addr, req.size);
          // debug code. retry 10 times
          int retrycnt = 0;
          while(ARCP2P_NO_ERROR != arcp2p_bar_attach(arc_handle, req.addr, req.size)) {
            retrycnt ++;
            printf("Bar attach failed, retry %d/10\n", retrycnt);
            if (retrycnt >= 10) {
              break;
            }
            arcp2p_bar_detach(arc_handle);
          }
          arcp2p_transfer(arc_handle, req.addr, req.offs, req.size, req.p_cpl, req.dir);
        }
      }
    }
    m2.unlock();
  }
}

void ARC_memory::Arcp2pSynchronize() {
  arcp2p_synchronize(arc_handle);
}

}}
