
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorRandom.cu"
#else

#include <ATen/cuda/CUDAContext.h>
#include <ATen/CUDAGenerator.h>
#include <ATen/Utils.h>
#include <utility>

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(renormRows)(struct THCState* state,
                             THCTensor* t) {
  THAssert(THCTensor_(nDimensionLegacyAll)(state, t) == 2);
  int64_t rows = THCTensor_(size)(state, t, 0);
  int64_t cols = THCTensor_(size)(state, t, 1);

  cudaDeviceProp* props = at::cuda::getCurrentDeviceProperties();
  THAssert(props != NULL);

  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;

  dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
  dim3 block(cols < maxThreads ? cols : maxThreads);

  renormRowsL1<scalar_t>
    <<<grid, block, block.x * sizeof(scalar_t),
    THCState_getCurrentStream(state)>>>(THCTensor_(data)(state, t),
                                        rows, cols);
}

void THCTensor_(multinomial)(struct THCState *state,
                              THCudaLongTensor *self,
                              at::Generator* gen_,
                              THCTensor *prob_dist,
                              int n_sample,
                              int with_replacement)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, prob_dist));
  auto gen = at::get_generator_or_default<at::CUDAGenerator>(gen_, at::cuda::detail::getDefaultCUDAGenerator());
  int inputSize = THCTensor_(nDimensionLegacyAll)(state, prob_dist);
  THArgCheck(inputSize > 0 && inputSize <= 2, 2,
             "prob_dist must be 1 or 2 dim");

  // Categories are in the innermost dimension
  int64_t numDist =
    inputSize == 1 ? 1 : THCTensor_(sizeLegacyNoScalars)(state, prob_dist, 0);
  int64_t numCategoriesLong =
    inputSize == 1 ? THCTensor_(sizeLegacyNoScalars)(state, prob_dist, 0) :
    THCTensor_(sizeLegacyNoScalars)(state, prob_dist, 1);

  // Since the index tensor is float, numCategories cannot exceed max
  // float integer precision
  THArgCheck(numCategoriesLong <= FLOAT32_MAX_CONSECUTIVE_INT, 2,
             "number of categories cannot exceed 2^24");
  int numCategories = (int) numCategoriesLong;

  THArgCheck(n_sample > 0, 3, "cannot sample <= 0 samples");

  if (!with_replacement) {
    THArgCheck(n_sample <= numCategories, 2,
               "cannot sample n_sample > prob_dist:size(1) samples without "
               "replacement");
  }

  int free_prob_dist = 0;

  // Restructure data for 2d
  if (inputSize == 1) {
    THCTensor *temp = THCTensor_(new)(state);
    THCTensor_(unsqueeze1d)(state, temp, prob_dist, 0);
    prob_dist = temp;
    free_prob_dist = 1;
  }

  THCudaLongTensor_resize2d(state, self, numDist, n_sample);

  // get current device properties
  cudaDeviceProp* props = at::cuda::getCurrentDeviceProperties();
  THAssert(props != NULL);
  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;
  int maxShared = props->sharedMemPerBlock;
  int requiredShared = (numCategories < maxThreads ? numCategories : maxThreads)
                                * (sizeof(scalar_t) + sizeof(accreal));

  if (n_sample == 1 && maxShared >= requiredShared) {
    // Optimized allocation-free implementation
    // To exploit greater parallelism for the sampling, generate the
    // Uniform random samples in a separate kernel launch, into
    // temporarily allocated memory. The device RNG is thread-limited
    THCTensor *sampled = THCTensor_(newWithSize2d)(state, numDist, n_sample);
    auto out = THTensor_wrap(sampled);
    at::native::uniform_cuda_(out, 0.0, 1.0, gen);

    dim3 block(numCategories < maxThreads ? numCategories : maxThreads);
    dim3 grid(numDist < numSM * 4 ? numDist : numSM * 4);

    sampleMultinomialOnce<scalar_t, accreal>
      <<<grid, block,
         requiredShared,
         THCState_getCurrentStream(state)>>>(
      THCudaLongTensor_data(state, self),
      numDist,
      numCategories,
      THCTensor_(data)(state, sampled),
      THCTensor_(data)(state, prob_dist),
      THCTensor_(stride)(state, prob_dist, 0),
      THCTensor_(stride)(state, prob_dist, 1)
      );
    THCTensor_(free)(state, sampled);
  } else {
    // Generic, slow implementation with memory allocations

    // For sampling without replacement, we modify the distribution
    // for subsequent samples in this space
    THCTensor* origDist = THCTensor_(new)(state);
    THCTensor_(resizeAs)(state, origDist, prob_dist);
    THCTensor_(copy)(state, origDist, prob_dist);

    THCTensor* normDist = THCTensor_(new)(state);
    THCTensor_(resizeAs)(state, normDist, prob_dist);

    THCTensor* prefixSum = THCTensor_(new)(state);

    // Renorm along rows
    THCTensor_(copy)(state, normDist, origDist);
    THCTensor_(renormRows)(state, normDist);

    // Prefix sum along rows
    THCTensor_(cumsum)(state, prefixSum, normDist, 1);
 
    std::pair<uint64_t, uint64_t> rng_engine_inputs;
    if (with_replacement) {
      {
        // See Note [Acquire lock when using random generators]
        std::lock_guard<std::mutex> lock(gen->mutex_);

        // each thread will utilize one random, however, since we have to use
        // curand_uniform4 (See Note [Register spilling in curand call for CUDA < 10]),
        // offset is 4.
        rng_engine_inputs = gen->philox_engine_inputs(4);
      }
      // Sample with replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from one
      // distribution concurrently.
      dim3 grid(numDist < MAX_NUM_BLOCKS ? numDist : MAX_NUM_BLOCKS);

      sampleMultinomialWithReplacement
        <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
          rng_engine_inputs,
          n_sample,
          THCudaLongTensor_data(state, self),
          numDist, numCategories,
          THCTensor_(data)(state, prefixSum),
          THCTensor_(data)(state, normDist));
    } else {
      // Sample without replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from a different
      // distribution concurrently.
      ptrdiff_t numBlocks = THCCeilDiv(numDist, (int64_t) 4);
      dim3 grid(numBlocks < MAX_NUM_BLOCKS ? numBlocks : MAX_NUM_BLOCKS);

      for (int sample = 0; sample < n_sample; ++sample) {
        if (sample > 0) {
          // Update probabilities
          // Renorm along rows
          THCTensor_(copy)(state, normDist, origDist);
          THCTensor_(renormRows)(state, normDist);

          // Prefix sum along rows
          THCTensor_(cumsum)(state, prefixSum, normDist, 1);
        }
        {
          // See Note [Acquire lock when using random generators]
          std::lock_guard<std::mutex> lock(gen->mutex_);
  
          // each thread will utilize one random, however, since we have to use
          // curand_uniform4 (See Note [Register spilling in curand call for CUDA < 10]),
          // offset is 4.
          rng_engine_inputs = gen->philox_engine_inputs(4);
        }

        // The kernel can only draw one sample before we have to
        // recalculate our distribution
        sampleMultinomialWithoutReplacement
          <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
            rng_engine_inputs,
            n_sample,
            sample,
            THCudaLongTensor_data(state, self),
            numDist, numCategories,
            THCTensor_(data)(state, origDist),
            THCTensor_(data)(state, prefixSum));
      }
    }

    THCTensor_(free)(state, prefixSum);
    THCTensor_(free)(state, normDist);
    THCTensor_(free)(state, origDist);
  }

  // Revert data restructuring based on input sizes
  if (inputSize == 1) {
    THCudaLongTensor_resize1d(state, self, n_sample);
  }
  if (free_prob_dist) {
    THCTensor_(free)(state, prob_dist);
  }
}

void THCTensor_(multinomialAliasSetup)(THCState *state, THCTensor *_probs, THCudaLongTensor *_J, THCTensor *_q){
  THArgCheck(_probs->dim() == 1, 1,
             "expected 1-D probability tensor, got %d-D probability tensor instead",
             _probs->dim());
  THAssert(THCTensor_(isContiguous)(state, _q));
  THAssert(THCudaLongTensor_isContiguous(state, _J));
  THCTensor *probs = THCTensor_(newContiguous)(state, _probs);
  THAssert(THCTensor_(isContiguous)(state, probs));
  int64_t inputsize = THCTensor_(nElement)(state, probs);
  THCudaLongTensor *smaller = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *larger = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *smaller_short = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *larger_short = THCudaLongTensor_newWithSize1d(state, inputsize);

  THCudaLongTensor_resize1d(state, _J, inputsize);
  THCTensor_(resize1d)(state, _q, inputsize);

  scalar_t one = ScalarConvert<int64_t, scalar_t>::to(1);
  int inputBlockDim = THCCeilDiv((int)inputsize + BLOCK_SIZE - 1, BLOCK_SIZE);
  aliasMultinomialFilter
    <<<inputBlockDim, BLOCK_SIZE, 0, THCState_getCurrentStream(state) >>>(
                     THCTensor_(data)(state, _q),
                     THCTensor_(data)(state, probs),
                     THCudaLongTensor_data(state, smaller),
                     THCudaLongTensor_data(state, larger),
                     THCudaLongTensor_data(state, _J),
                     THCudaLongTensor_data(state, smaller_short),
                     THCudaLongTensor_data(state, larger_short),
                     one, inputsize
                     );

  THCudaLongTensor_nonzero(state, smaller_short, smaller);
  THCudaLongTensor_nonzero(state, larger_short, larger);
  int h_large_c = THCudaLongTensor_nElement(state, larger_short);
  THCudaLongTensor_resize1d(state, smaller_short, inputsize);
  THCudaLongTensor_resize1d(state, larger_short, inputsize);
  aliasMultinomialSetup
    <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
                THCudaLongTensor_data(state, _J),
                THCTensor_(data)(state, _q),
                inputsize,
                THCudaLongTensor_data(state, smaller_short),
                THCudaLongTensor_data(state, larger_short),
                inputsize - h_large_c, h_large_c
                );
  scalar_t q_max = THCTensor_(maxall)(state, _q);
  condDiv<<<
    inputBlockDim, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
                      THCTensor_(data)(state, _q),
                      THCudaLongTensor_data(state, _J),
                      inputsize, q_max
                      );

  THCudaLongTensor_free(state, smaller);
  THCudaLongTensor_free(state, larger);
  THCudaLongTensor_free(state, smaller_short);
  THCudaLongTensor_free(state, larger_short);
  THCTensor_free(state, probs);
}

void THCTensor_(multinomialAliasDraw)(THCState *state, THCudaLongTensor *self, at::Generator* gen_, THCTensor *_q, THCudaLongTensor *_J, int n_sample){
  THArgCheck(_q->dim() == 1, 1,
             "expected 1-D probability table, got %d-D probability table instead",
             _q->dim());
  THArgCheck(_J->dim() == 1, 2,
             "expected 1-D alias table, got %d-D alias table instead",
             _J->dim());
  THArgCheck(n_sample > 0, 3, "cannot sample <= 0 samples");
  THAssert(THCTensor_(isContiguous)(state, _q));
  THAssert(THCudaLongTensor_isContiguous(state, _J));
  auto gen = at::get_generator_or_default<at::CUDAGenerator>(gen_, at::cuda::detail::getDefaultCUDAGenerator());
  int64_t K = THCudaLongTensor_nElement(state, _J);
  THCudaLongTensor_resize1d(state, self, n_sample);
  ptrdiff_t size = THCudaLongTensor_nElement(state, self);

  THCTensor *uniform = THCTensor_(newWithSize1d)(state, n_sample);
  THCTensor *bernoulli = THCTensor_(newWithSize1d)(state, n_sample);

  auto out_uniform = THTensor_wrap(uniform);
  auto out_bernoulli = THTensor_wrap(bernoulli);
  at::native::uniform_cuda_(out_uniform, 0, K, gen);
  at::native::uniform_cuda_(out_bernoulli, 0, 1, gen);

  multinomialAliasDrawKernel
    <<<THCCeilDiv((int)n_sample+BLOCK_SIZE-1, BLOCK_SIZE), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
          size,
          THCudaLongTensor_data(state, self),
          THCudaLongTensor_data(state, _J),
          THCTensor_(data)(state, _q),
          K,
          THCTensor_(data)(state, uniform),
          THCTensor_(data)(state, bernoulli)
          );
  THCTensor_(free)(state, uniform);
  THCTensor_(free)(state, bernoulli);
}

#endif
#endif
